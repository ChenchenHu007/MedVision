#include "roi_align_rotated_3d_cuda_kernel.cuh"

using namespace at;

void ROIAlignRotated3DForwardCUDAKernelLauncher(
    const at::Tensor features, const at::Tensor rois, const float spatial_scale,
    const int sampling_ratio, const bool aligned, const int order,
    const int channels,
    const int depth, const int height, const int width,
    const int num_rois,
    const int pooled_depth, const int pooled_height, const int pooled_width,
    at::Tensor output) {
  const int output_size = num_rois * pooled_depth * pooled_height * pooled_width * channels;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.type(), "ROIAlignRotated3DLaucherForward", ([&] {
        const scalar_t *bottom_data = features.contiguous().data<scalar_t>();
        const scalar_t *rois_data = rois.contiguous().data<scalar_t>();
        scalar_t *top_data = output.contiguous().data<scalar_t>();

        roi_align_rotated_3d_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, bottom_data, rois_data, scalar_t(spatial_scale),
                sampling_ratio, aligned, order, channels,
                depth, height, width,
                pooled_depth, pooled_height, pooled_width,
                top_data);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}

void ROIAlignRotated3DBackwardCUDAKernelLauncher(
    const at::Tensor top_grad, const at::Tensor rois, const float spatial_scale,
    const int sampling_ratio, const bool aligned, const int order,
    const int channels,
    const int depth, const int height, const int width,
    const int num_rois,
    const int pooled_depth, const int pooled_height, const int pooled_width,
    at::Tensor bottom_grad) {
  const int output_size = num_rois * pooled_depth * pooled_height * pooled_width * channels;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.type(), "ROIAlignRotated3DLaucherBackward", ([&] {
        const scalar_t *top_diff = top_grad.data<scalar_t>();
        const scalar_t *rois_data = rois.contiguous().data<scalar_t>();
        scalar_t *bottom_diff = bottom_grad.data<scalar_t>();
        roi_align_rotated_3d_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, top_diff, rois_data, spatial_scale, sampling_ratio,
                aligned, order, channels,
                depth, height, width,
                pooled_depth, pooled_height, pooled_width,
                bottom_diff);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}
