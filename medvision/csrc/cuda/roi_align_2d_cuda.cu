#include "roi_align_2d_cuda_kernel.cuh"

using namespace at;

void ROIAlign2DForwardCUDAKernelLauncher(
    const at::Tensor features, const at::Tensor rois, const float spatial_scale,
    const int sampling_ratio, const int order,
    const int channels,
    const int height, const int width,
    const int num_rois,
    const int pooled_height, const int pooled_width,
    at::Tensor output) {
  const int output_size = num_rois * pooled_height * pooled_width * channels;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.type(), "ROIAlign2DLauncherForward", ([&] {
        const scalar_t *bottom_data = features.contiguous().data<scalar_t>();
        const scalar_t *rois_data = rois.contiguous().data<scalar_t>();
        scalar_t *top_data = output.contiguous().data<scalar_t>();

        roi_align_2d_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, bottom_data, rois_data, scalar_t(spatial_scale),
                sampling_ratio, order, channels,
                height, width,
                pooled_height, pooled_width,
                top_data);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}

void ROIAlign2DBackwardCUDAKernelLauncher(
    const at::Tensor top_grad, const at::Tensor rois, const float spatial_scale,
    const int sampling_ratio, const int order,
    const int channels,
    const int height, const int width,
    const int num_rois,
    const int pooled_height, const int pooled_width,
    at::Tensor bottom_grad) {
  const int output_size = num_rois * pooled_height * pooled_width * channels;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.type(), "ROIAlign2DLauncherBackward", ([&] {
        const scalar_t *top_diff = top_grad.data<scalar_t>();
        const scalar_t *rois_data = rois.contiguous().data<scalar_t>();
        scalar_t *bottom_diff = bottom_grad.data<scalar_t>();
        roi_align_2d_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, top_diff, rois_data, spatial_scale, sampling_ratio,
                order, channels,
                height, width,
                pooled_height, pooled_width,
                bottom_diff);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}
